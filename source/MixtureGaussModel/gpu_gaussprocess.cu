#include "hip/hip_runtime.h"
#include "gpu_gaussprocess_cu.h"

#include <cutil_inline.h>
#include <iostream>
#include <ctime>



__global__ static void cuda_gmm_calcDiff_impl(float* diff, const float* value, const float* mean, const int* size)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int i = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(i < *size)
	{
		diff[i] = abs(value[i] - mean[i]);
	}
}

__global__ static void cuda_gmm_updatePara_impl(const float* value, const float* diff, float* weight, float* mean, float* stdDev, float*p)
{
}



__global__ static void cuda_gmm_match_impl(const float* value, float* diff, float* weight, float* mean, float* stdDev, float* p, bool* match, const int *len, const int* num, const float *alpha, const float* compAlpha)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int i = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(i < *len * *num)
	{
		diff[i] = abs(value[i] - mean[i]);
		
		if(diff[i] <= 4 * stdDev[i])
		{
			match[i] = true;
			
			weight[i] = *compAlpha * weight[i] + *alpha;

			p[i] = 1 / (pow(2 * 3.141, 1.5) * stdDev[i] ) * exp( -0.5 * pow((double)(value[i] - stdDev[i]), 2.0) / pow((double)stdDev[i], 2.0));

			mean[i] = (1.0 - p[i]) * mean[i] + p[i] * value[i];

			stdDev[i] = sqrt((1.0 - p[i]) * pow(stdDev[i], 2 ) + p[i] * pow(value[i] - mean[i], 2));
		}
		else
		{
			weight[i] *= *compAlpha;
		}
	}
	__syncthreads();
}




__global__ static void cuda_gmm_normalize_impl(float* weight, float* weightSum, const int *len, const int* n)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int i = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(i < *len)
	{
		weightSum[i] = 0.0f;
		for(int j = 0; j < *n; ++j)
		{
			weightSum[i] += weight[i * *n + j];
		}
	}

	__syncthreads();

	if(i < *len)
	{
		for(int j = 0; j < *n; ++j)
		{
			weight[i + j] /= weightSum[i];
		}
	}

}

__global__ static void cuda_gmm_calcCmpFactor_impl(const float* weight, const float* stdDev, float* cmpFactor, const int* len, const int* num)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int i = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(i < *len * *num)
	{
		cmpFactor[i] = weight[i] / stdDev[i];
	}
}




void cudaGMMMatch(const float* value, float* weight, float* mean, float* stdDev, float* p,
				  bool* match, const int len, const int num, const float alpha)
{
	assert(value || mean || stdDev || p || match);

	dim3 blockD(512, 512);
	dim3 threadD(16, 32);


	float compAlpha = 1.0f - alpha;
	
	float* d_value = NULL;
	float* d_diff = NULL;
	float* d_weight = NULL;
	float* d_mean = NULL;
	float* d_stdDev = NULL;
	float* d_p = NULL;
	bool* d_match = NULL;
	int* d_len = NULL;
	int* d_num = NULL;
	float* d_alpha = NULL;
	float* d_compAlpha = NULL;

	float* d_weightSum = NULL;

	for(int i = 0; i < len * num; ++i)
	{
		match[i] = false;
	}

	//to allocate memory on device
	cutilSafeCall(hipMalloc((void**)&d_value, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_diff, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_weight, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_mean, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_stdDev, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_p, sizeof(float) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_match, sizeof(bool) * len * num));
	cutilSafeCall(hipMalloc((void**)&d_len, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_num, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_alpha, sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_compAlpha, sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_weightSum, sizeof(float) * len));

	//copy data from host to device
	cutilSafeCall(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_num, &num, sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_compAlpha, &compAlpha, sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_match, match, sizeof(bool) * len * num, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_value, value, sizeof(float) * len * num, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_weight, weight, sizeof(float) * len * num, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_mean, mean, sizeof(float) * len * num, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_stdDev, stdDev, sizeof(float) * len * num, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_p, p, sizeof(float) * len * num, hipMemcpyHostToDevice));

	cuda_gmm_match_impl<<<blockD, threadD, 0>>>(d_value, d_diff, d_weight, d_mean, d_stdDev, d_p, d_match, d_len, d_num, d_alpha, d_compAlpha);

	//cuda_gmm_normalize_impl<<<blockD, threadD, 0>>>(d_weight, d_weightSum, d_len, d_num);
	
	cutilSafeCall(hipMemcpy(match, d_match, sizeof(bool) * len * num, hipMemcpyDeviceToHost));

	//copy back
	cutilSafeCall(hipMemcpy(weight, d_weight, sizeof(float) * len * num, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(mean, d_mean, sizeof(float) * len * num, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(stdDev, d_stdDev, sizeof(float) * len * num, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(p, d_p, sizeof(float) * len * num, hipMemcpyDeviceToHost));

		
	//free 
	cutilSafeCall(hipFree(d_value));
	cutilSafeCall(hipFree(d_diff));
	cutilSafeCall(hipFree(d_weight));
	cutilSafeCall(hipFree(d_mean));
	cutilSafeCall(hipFree(d_stdDev));
	cutilSafeCall(hipFree(d_p));
	cutilSafeCall(hipFree(d_match));
	cutilSafeCall(hipFree(d_len));
	cutilSafeCall(hipFree(d_num));
	cutilSafeCall(hipFree(d_alpha));
	cutilSafeCall(hipFree(d_compAlpha));

	cutilSafeCall(hipFree(d_weightSum));

	d_value = NULL;
	d_weight = NULL;
	d_mean = NULL;
	d_stdDev = NULL;
	d_p = NULL;
	d_match = NULL;
	d_len = NULL;
	d_num = NULL;
	d_alpha = NULL;
	d_compAlpha = NULL;
	d_weightSum = NULL;

}



/*
**to calculate the gauss value in device
*/
__global__ void cuda_gauss_impl(float* result, const float* value, const float* mean, float* stdDev, const int* len)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int index = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(index < *len)
		result[index] = 1.0f / (pow(2.0f * 3.141f, 1.5f) * stdDev[index] ) *
		exp( -0.5f * pow((double)(value[index] - stdDev[index]), 2.0) / pow((double)stdDev[index], 2.0));

	__syncthreads();
}

void cudaGauss(float* result, const float* value, const float* mean, const float* stdDev, const int len)
{
	assert(mean && value);

	dim3 blockD(512, 512);
	dim3 threadD(16, 32);

	float* d_result = NULL;
	float* d_value = NULL;
	float* d_stdDev = NULL;
	float* d_mean = NULL;
	int* d_len;

	unsigned int timer = 0;

	cutilSafeCall(hipMalloc((void**)&d_result, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_value, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_stdDev, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_mean, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_len, sizeof(int)));

	cutilSafeCall(hipMemcpy(d_value, value, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_stdDev, stdDev, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_mean, mean, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice));

	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutStartTimer(timer));

	cuda_gauss_impl<<<blockD, threadD, 0>>>(d_result, d_value, d_mean, d_stdDev, d_len);
	cutilCheckError(cutStopTimer(timer));

	//std::cout << "gpu time: " << cutGetTimerValue(timer) <<  std::endl;

	cutilSafeCall(hipMemcpy(result, d_result, sizeof(float) * len, hipMemcpyDeviceToHost));

	cutilSafeCall(hipFree(d_result));
	cutilSafeCall(hipFree(d_stdDev));
	cutilSafeCall(hipFree(d_mean));
	cutilSafeCall(hipFree(d_value));
	cutilSafeCall(hipFree(d_len));

	d_result = NULL;
	d_stdDev = NULL;
	d_mean = NULL;
	d_value = NULL;
	d_len = NULL;

}

__global__ void cuda_diff_impl(const float* src1, const float* src2, float* dst, const int* len)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int index = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(index < *len)
		dst[index] = src1[index] - src2[index];
}

void cudaDiff(const float* src1, const float* src2, float* dst, const int len)
{
	assert(src1 && src2 && dst);

	dim3 blockD(512, 512);
	dim3 threadD(16, 32);

	float* d_src1 = NULL;
	float* d_src2 = NULL;
	float* d_dst = NULL;
	int* d_len;

	unsigned int timer = 0;

	cutilSafeCall(hipMalloc((void**)&d_src1, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_src2, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_dst, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_len, sizeof(int)));

	cutilSafeCall(hipMemcpy(d_src1, src1, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_src2, src2, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice));

	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutStartTimer(timer));

	cuda_diff_impl<<<blockD, threadD, 0>>>(d_src1, d_src2, d_dst, d_len);
	cutilCheckError(cutStopTimer(timer));

	std::cout << "gpu time: " << cutGetTimerValue(timer) <<  std::endl;

	cutilSafeCall(hipMemcpy(dst, d_dst, sizeof(float) * len, hipMemcpyDeviceToHost));

	cutilSafeCall(hipFree(d_src1));
	cutilSafeCall(hipFree(d_src2));
	cutilSafeCall(hipFree(d_dst));
	cutilSafeCall(hipFree(d_len));

	d_src1 = NULL;
	d_src2 = NULL;
	d_dst = NULL;

}


__global__ void cuda_updateP_impl(float* p, const float* weight, const float* gaussValue, const int* len, const float* alpha)
{
	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int index = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;

	if(index < *len)
	{
		p[index] = *alpha * weight[index] * gaussValue[index];
	}
}


void cudaUpdateP(float* p, const float* weight, const float* gaussValue, const int len, const float alpha)
{

	assert(weight && gaussValue);

	dim3 blockD(512, 512);
	dim3 threadD(16, 32);

	float* d_p = NULL;
	float* d_weight = NULL;
	float* d_gauss = NULL;
	int* d_len = NULL;
	float* d_alpha = NULL;

	unsigned int timer = 0;

	cutilSafeCall(hipMalloc((void**)&d_p, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_weight, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_gauss, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_len, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_alpha, sizeof(float)));

	cutilSafeCall(hipMemcpy(d_weight, weight, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_gauss, gaussValue, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice));
	

	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutStartTimer(timer));

	cuda_updateP_impl<<<blockD, threadD, 0>>>(d_p, d_weight, d_gauss, d_len, d_alpha);
	cutilCheckError(cutStopTimer(timer));

	std::cout << "gpu time: " << cutGetTimerValue(timer) <<  std::endl;

	cutilSafeCall(hipMemcpy(p, d_p, sizeof(float) * len, hipMemcpyDeviceToHost));

	cutilSafeCall(hipFree(d_p));
	cutilSafeCall(hipFree(d_weight));
	cutilSafeCall(hipFree(d_gauss));
	cutilSafeCall(hipFree(d_len));
	cutilSafeCall(hipFree(d_alpha));


	d_p = NULL;
	d_weight = NULL;
	d_gauss = NULL;
	d_len = NULL;
	d_alpha = NULL;
}




__global__ void cuda_match_impl(const float* value, float* mean,
								float* stdDev, const int* len, const float* alpha, unsigned char* fg)
{

	int blx = blockIdx.x;
	int bly = blockIdx.y;
	int thx = threadIdx.x;
	int thy = threadIdx.y;

	int i = (bly * blockDim.y + thy) * blockDim.x * gridDim.x + blx * blockDim.x + thx;


	
	if(i < *len)
	{
		if(abs(mean[i] - value[i]) < 3.0f * stdDev[i])
		{
			
			mean[i] = (1.0f - *alpha) * mean[i] + *alpha * value[i];
			stdDev[i] = sqrt((1.0f - *alpha) * stdDev[i] * stdDev[i] + *alpha * (value[i] - mean[i]) * (value[i] - mean[i]));
			fg[i] = 0;
		}
		else
		{
			fg[i] = 255;
			mean[i] = value[i];
			stdDev[i] = 8;
		}
	}
	__syncthreads();
}


void cudaMatch(const float* value, float* mean, float* stdDev, const int len, const float alpha, unsigned char* fg)
{
	float* d_value = NULL;
	float* d_mean = NULL;
	float* d_stdDev = NULL;
	int* d_len = NULL;
	float* d_alpha = NULL;
	unsigned char* d_fg = NULL;

	unsigned int timer = 0;

	dim3 blockD(352, 288);
	dim3 threadD(32, 16);

	cutilSafeCall(hipMalloc((void**)&d_value, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_mean, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_stdDev, sizeof(float) * len));
	cutilSafeCall(hipMalloc((void**)&d_len, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_alpha, sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&d_fg, sizeof(unsigned char) * len));

	cutilSafeCall(hipMemcpy(d_value, value, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_mean, mean, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_stdDev, stdDev, sizeof(float) * len, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice));


	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutStartTimer(timer));
	cuda_match_impl<<<blockD, threadD, 0>>>(d_value, d_mean, d_stdDev, d_len, d_alpha, d_fg);

	hipDeviceSynchronize();
	cutilCheckError(cutStopTimer(timer));

	//std::cout << "gpu time: " << cutGetTimerValue(timer) <<  std::endl;

	cutilSafeCall(hipMemcpy(mean, d_mean, sizeof(float) * len, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(stdDev, d_stdDev, sizeof(float) * len, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(fg, d_fg, sizeof(unsigned char) * len, hipMemcpyDeviceToHost));

	cutilSafeCall(hipFree(d_value));
	cutilSafeCall(hipFree(d_mean));
	cutilSafeCall(hipFree(d_stdDev));
	cutilSafeCall(hipFree(d_len));
	cutilSafeCall(hipFree(d_alpha));
	cutilSafeCall(hipFree(d_fg));

	d_value = NULL;
	d_mean = NULL;
	d_stdDev = NULL;
	d_len = NULL;
	d_alpha = NULL;
	d_fg = NULL;

}

void cudaRun(const float* value, float* mean, float* stdDev, const int len, const float alpha, unsigned char* fg)
{
	cudaMatch(value, mean, stdDev, len, alpha, fg);
}